#include "hip/hip_runtime.h"
#include <sys/stat.h>
#include <stdlib.h>
#include <stdio.h>
#include <jpeglib.h>
#include <jerror.h>
//config.h contains a number of needed definitions
#include "config.h"  
#include "histogram.h"
#include "wrappers.h"
#include "h_classify.h"
#include "d_classify.h"

#include "models.h"

//prototypes for functions in this file 
static void parseCommandArgs(int, char **, char *);
static void printUsage();
static void readJPGImage(char *, unsigned char **, int *, int *);
static void compareHistograms(histogramT *, histogramT *, int);
static void compareComparisons(comparisonT *, comparisonT *, int);
static void initHistogram(char *, histogramT *, int);
static void printTopTwo(comparisonT *);

/*
    main 
    Opens the jpg file and reads the contents.  Uses the CPU
    to build a histogram of the image, optionally outputting
    the histogram to a file in the form of a C struct initialization.  
    If the save option is not provided the program will also classify
    the image on the CPU and histogram and classify the image on the GPU.
    It compares the CPU and GPU results to make sure they match
    and outputs the times it takes on the CPU and the GPU to build the 
    histogram and perform the classification.
*/
int main(int argc, char * argv[])
{
    unsigned char * hPin, *dPin; 
    histogramT * h_hgram, * d_hgram;
    char imagefile[NAMELEN];
    int width, height;
    float cpuTime, gpuTime;

    //need an array of these; one for each model
    //one array for the GPU and one array for the CPU
    comparisonT * hresult = (comparisonT *) Malloc(sizeof(comparisonT) * MODELS);
    comparisonT * dresult = (comparisonT *) Malloc(sizeof(comparisonT) * MODELS);

    parseCommandArgs(argc, argv, imagefile);

    //create histogram structs for the host and the device
    h_hgram = (histogramT *) Malloc(sizeof(histogramT));
    d_hgram = (histogramT *) Malloc(sizeof(histogramT));
    initHistogram(imagefile, h_hgram, TOTALBINS);


    //read the image for the CPU
    readJPGImage(imagefile, &hPin, &width, &height);
    printf("\nComputing histogram and classifying %s.\n", imagefile);

    //use the CPU to build the histogram and classify it
    cpuTime = h_classify(h_hgram, hresult, models, MODELS, hPin, height, width); 
    printf("\tCPU time: \t\t%f msec\n", cpuTime);

    //read the image for the GPU
    readJPGImage(imagefile, &dPin, &width, &height);

    //use the GPU to build the histogram and classify it
    initHistogram(imagefile, d_hgram, TOTALBINS);
    gpuTime = d_classify(d_hgram, dresult, models, MODELS, dPin, height, width);
    compareHistograms(d_hgram, h_hgram, TOTALBINS);
    compareComparisons(dresult, hresult, MODELS);
    printf("\tGPU time: \t\t%f msec\n", gpuTime);
    printf("\tSpeedup: \t\t%f\n", cpuTime/gpuTime);
    printTopTwo(dresult);

    free(d_hgram);
    free(h_hgram);
    free(hPin);
    free(dPin);
    return EXIT_SUCCESS;
}

/*
    initHistogram
    Initializes a histogram struct by setting the bin values to 0 and setting
    the fileName field to the name of the file containing the image to histogram.
*/
void initHistogram(char * fileName, histogramT * histP, int length)
{
    int i;
    strncpy(histP->fileName, fileName, sizeof(histP->fileName));
    for (i = 0; i < length; i++)
    {
       histP->histogram[i] = 0;
    }
}

/*
    printTopTwo
    Finds and prints the top two matches in the comparison struct.
    comparison values will range from 0 to 1.
    An exact match will have a comparison value of 1.0, which indicates
    the model matches the input image exactly.  
    A comparison value of .8 means that %80 of the pixels in the
    model and the input image are the same.
*/
   
void printTopTwo(comparisonT * result)
{
    int first = -1, second = -1;
    int i;
    for (i = 0; i < MODELS; i++)
    {
        if (first == -1)  //both first and second are -1
            first = i;
        else if (second == -1) //first is not -1
        {
            if (result[i].comparison > result[first].comparison)
            {
                second = first;
                first = i;
            } else
            {
                second = i;
            }
        } else if (result[i].comparison > result[first].comparison)
        {
            second = first;
            first = i;
        } else if (result[i].comparison > result[second].comparison)
        {
            second = i;
        }
    }
    printf("\nMatches\n");
    printf("-------\n");
    printf("\tFirst:  %s    \t%5.1f%%\n", result[first].fileName, 
           (result[first].comparison * 100));
    printf("\tSecond: %s    \t%5.1f%%\n", result[second].fileName, 
           (result[second].comparison * 100));
}

/* 
    compareHistograms
    This function takes two histogramT structs. One histogramT 
    contains bins calculated  by the GPU. The other histogramT
    contains bins calculated by the CPU. This function examines
    each bin to see that they match.

    d_Pout - histogram calculated by GPU
    h_Pout - histogram calculated by CPU
    length - number of bins in histogram
    
    Outputs an error message and exits program if the histograms differ.
*/
void compareHistograms(histogramT * d_Pout, histogramT * h_Pout, int length)
{
    int i;
    for (i = 0; i < length; i++)
    {
        if (d_Pout->histogram[i] != h_Pout->histogram[i])
        {
            printf("Histograms don't match.\n");
            printf("host bin[%d] = %d\n", i, h_Pout->histogram[i]);
            printf("device bin[%d] = %d\n", i, d_Pout->histogram[i]);
            exit(EXIT_FAILURE);
        }
    }
}

/* 
    compareComparisons
    This function takes two comparisonT structs. One comparisonT 
    contains a comparison array calculated  by the GPU.  The other 
    comparsionT contains a comparison array calculated
    by the CPU.  This function examines each comparison array
    element to see that they match.

    d_Pout - comparison calculated by GPU
    h_Pout - comparison calculated by CPU
    length - number of comparison
    
    Outputs an error message and exits program if the comparisons differ.
*/
void compareComparisons(comparisonT * d_Pout, comparisonT * h_Pout, int length)
{
    int i;
    for (i = 0; i < length; i++)
    {
        if (abs(d_Pout[i].comparison - h_Pout[i].comparison) > 0.01)
        {
            printf("Comparisons don't match for %s.\n", d_Pout[i].fileName);
            printf("host comparison[%d] = %f\n", i, h_Pout[i].comparison);
            printf("device comparison[%d] = %f\n", i, d_Pout[i].comparison);
            exit(EXIT_FAILURE);
        }
    }
}

/*
    readJPGImage
    This function opens a jpg file and reads the contents.  
    
    The array Pin is initialized to the pixel bytes.  width and height
    are pointers to ints that are set to those values.
    filename is the name of the .jpg file
*/
void readJPGImage(char * filename, unsigned char ** Pin,
                  int * width, int * height)
{
   unsigned long dataSize;             // length of the file
   int channels;                       //  3 =>RGB   4 =>RGBA 
   unsigned char * rowptr[1];          // pointer to an array
   unsigned char * jdata;              // data for the image
   struct jpeg_decompress_struct info; //for our jpeg info
   struct jpeg_error_mgr err;          //the error handler

   FILE * fp = fopen(filename, "rb"); //read binary
   if (fp == NULL)
   {
      fprintf(stderr, "Error reading file %s\n", filename);
      printUsage();
   }

   info.err = jpeg_std_error(& err);
   jpeg_create_decompress(&info);

   jpeg_stdio_src(&info, fp);
   jpeg_read_header(&info, TRUE);   // read jpeg file header
   jpeg_start_decompress(&info);    // decompress the file
   //set width and height
   (*width) = info.output_width;
   (*height) = info.output_height;
   channels = info.num_components;
   if (channels != CHANNELS)
   {
      fprintf(stderr, "%s is not an RGB jpeg image\n", filename);
      printUsage();
   }

   dataSize = (*width) * (*height) * channels;
   jdata = (unsigned char *)malloc(dataSize);
   if (jdata == NULL) fprintf(stderr, "Fatal error: malloc failed\n");
   while (info.output_scanline < info.output_height) // loop
   {
      // Enable jpeg_read_scanlines() to fill our jdata array
      rowptr[0] = (unsigned char *)jdata +  // secret to method
                  channels * info.output_width * info.output_scanline;

      jpeg_read_scanlines(&info, rowptr, 1);
   }
   jpeg_finish_decompress(&info);   //finish decompressing
   jpeg_destroy_decompress(&info);
   fclose(fp);                      //close the file
   (*Pin) = jdata;
   return;
}


/*
    parseCommandArgs
    This function parses the command line arguments. The program is executed 
    like this:
    ./classify <file>.jpg
    In addition, it checks to see if the last command line argument
    is a jpg file and sets imageFile to argv[1] when argv[1] is the name of the jpg
    file.  
*/
void parseCommandArgs(int argc, char * argv[], char imageFile[NAMELEN])
{
    struct stat buffer;
    if (argc < 2 || strncmp("-h", argv[1], 3) == 0) 
    {
        printUsage();
    } 

    //check the input file name (must end with .jpg)
    int len = strlen(argv[1]);
    if (len < 5) printUsage();
    if (strncmp(".jpg", &argv[1][len - 4], 4) != 0) printUsage();

    //stat function returns 1 if file does not exist
    if (stat(argv[1], &buffer)) printUsage();
    strcpy(imageFile, argv[1]);
}

/*
    printUsage
    This function is called if there is an error in the command line
    arguments or if the .jpg file that is provided by the command line
    argument is improperly formatted.  It prints usage information and
    exits.
*/
void printUsage()
{
    printf("This application takes as input the name of a .jpg\n");
    printf("file containing a color image and creates a histogram\n");
    printf("of the image. It then computes an intersection of this histogram\n");
    printf("and the other histograms defined in 'models.h'. It outputs the\n");
    printf("names of the two best matching images. This work is\n");
    printf("performed on the CPU and the GPU. Their results are timed and\n");
    printf("compared.\n");
    printf("\nusage: ./classify <name>.jpg\n");
    printf("       <name>.jpg is the name of the input jpg file.\n");
    printf("Examples:\n");
    printf("./classify images/WonderWoman1.jpg\n");
    exit(EXIT_FAILURE);
}
