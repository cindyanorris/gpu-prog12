#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "config.h"
#include "histogram.h"
#include "d_classify.h"
#include "CHECK.h"
#include "wrappers.h"

//parameters for building the histogram from the image
//TILEWIDTH is number of pixels in a row that a single thread will handle
#define TILEWIDTH 8 
#define HISTBLOCKDIM 32

//parameter is for classifying the image
#define CLASSBLOCKDIM 128

//prototypes for functions local to this file
static float histogramOnGPU(histogramT *, unsigned char *, int, int);
static float classifyOnGPU(float *, int *, int modelCt);

//prototypes for the kernels
static __global__ void d_histoKernel(histogramT *, unsigned char *, int, int, int);
static __global__ void d_classifyKernel(float *, float *, int *);
static __global__ void emptyKernel();

//prototypes of functions called by d_classifyKernel
static __device__ void computeHistSz(int *, int *);
static __device__ void normalizeHist(float *, int *, int);
static __device__ void intersection(float * normHistograms, float * intersect);

//for debugging
static __device__ void printFloatArray(float * array, int startIdx, int length);

/*
    d_classify
    Performs image classification on the GPU by first building a histogram
    to represent the image and then comparing the histogram to each of the
    histogram models.

    Outputs:
    Phisto - pointer to histogramT struct containing the bins 
    dresult - comparisonT array of structs; one element per model

    Inputs:
    models - an array of pointers to histogramT structs; one element per
             model to be compared to the input
    Pin - array contains the color pixels of the image to be used for 
          building a histogram and doing the classification
    width and height - dimensions of the image
 
    Returns the amount of time it takes to build the histogram and
      classify the image
*/
float d_classify(histogramT * Phisto, comparisonT * dresult, 
                 histogramT ** models, int modelCt, unsigned char * Pin,
                 int height, int width) 
{
    //THIS CODE IS COMPLETE

    float gpuMsecTime1, gpuMsecTime2;

    //launch an empty kernel to get more accurate timing
    emptyKernel<<<1024, 1024>>>();

    //build a histogram of the input image
    gpuMsecTime1 = histogramOnGPU(Phisto, Pin, height, width);

    //allocate array to hold all histograms, including the histogram for the input
    int * histograms = (int *) Malloc(sizeof(int) * (modelCt + 1) * TOTALBINS);

    //copy the histogram for the input to the beginning of the array
    memcpy(histograms, Phisto->histogram, sizeof(int) * TOTALBINS);

    //copy the remaining histograms
    for (int i = 1; i <= modelCt; i++) 
        memcpy(&histograms[i*TOTALBINS], models[i - 1]->histogram, sizeof(int) * TOTALBINS);

    //allocate an array of floats to hold the comparisons
    float * comparisons = (float *) Malloc(sizeof(int) * modelCt);

    //perform the classification
    gpuMsecTime2 = classifyOnGPU(comparisons, histograms, modelCt);

    //copy the results into the output
    for (int i = 0; i < modelCt; i++)
    {
        dresult[i].comparison = comparisons[i];
        strncpy(dresult[i].fileName, models[i]->fileName, NAMELEN);
    }

    return gpuMsecTime1 + gpuMsecTime2;
}

/*
   histogramOnGPU
   Builds a histogram to represent the input image.

   Outputs:
   Phisto - pointer to the histogramT struct containing the bins

   Inputs:
   Pin - array contains the color pixels of the image to be used for 
         building a histogram
   width and height -  dimensions of the image
   pitch - size of each row
 
   Returns the amount of time it takes to build the histogram 
*/
float histogramOnGPU(histogramT * Phisto, unsigned char * Pin, int height, 
                     int width)
{
    //THIS CODE IS COMPLETE

    hipEvent_t start_gpu, stop_gpu;
    float gpuMsecTime = -1;

    unsigned char * d_Pin;
    int pitch;
    histogramT * d_Phisto;

    //create an array on the GPU to hold the pitched image
    CHECK(hipMallocPitch((void **)&d_Pin, (size_t *) &pitch,
                          (size_t) (width * CHANNELS),
                          (size_t) height));
    for (int i = 0; i < height; i++)
       CHECK(hipMemcpy(&d_Pin[i * pitch], &Pin[i * width * CHANNELS],
             width * CHANNELS, hipMemcpyHostToDevice));

    //create the array on the GPU to hold the histogram
    CHECK(hipMalloc((void **)&d_Phisto, sizeof(histogramT)));
    CHECK(hipMemcpy(d_Phisto, Phisto, sizeof(histogramT),
          hipMemcpyHostToDevice));

    //Use cuda functions to do the timing 
    //create event objects
    CHECK(hipEventCreate(&start_gpu));
    CHECK(hipEventCreate(&stop_gpu));
    
    //build the histogram
    CHECK(hipEventRecord(start_gpu));

    //each thread calculates TILEWIDTH elements in a row
    dim3 grid(ceil(width/(float)(HISTBLOCKDIM * TILEWIDTH)),
              ceil(height/(float)HISTBLOCKDIM), 1);
    dim3 block(HISTBLOCKDIM, HISTBLOCKDIM, 1);

    d_histoKernel<<<grid, block>>>(d_Phisto, d_Pin, height, width, pitch);

    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipMemcpy(Phisto, d_Phisto, sizeof(histogramT),
          hipMemcpyDeviceToHost));
    //record the ending time and wait for event to complete
    CHECK(hipEventSynchronize(stop_gpu));
    //calculate the elapsed time between the two events 
    CHECK(hipEventElapsedTime(&gpuMsecTime, start_gpu, stop_gpu));
    //CHECK(hipEventDestroy(start_gpu));
    //CHECK(hipEventDestroy(stop_gpu));

    return gpuMsecTime;
}

/*
   d_histoKernel
   Kernel code executed by each thread on its own data when the kernel is
   launched. Each thread operates on TILEWIDTH pixels in a row.

   Inputs:
   Pin - array contains the color pixels to be used to build the histogram
   width and height - dimensions of the image
   pitch - size of each row

   Output:
   histo - pointer to a histogramT struct that contains an array of bins
*/
__global__
void d_histoKernel(histogramT * histo, unsigned char * Pin, int height,
                  int width, int pitch)
{
    //THIS CODE IS COMPLETE.  You can replace it with a faster version
    //if you like, but the shared memory version won't work with all
    //TOTALBINS sizes.  If you use that one, the largest BIN value can
    //only be 8.

    int colStart = (blockIdx.x * blockDim.x + threadIdx.x) * TILEWIDTH;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col;

    //use a privatization technique to reduce the number of atomic adds
    int accumulator = 0;
    int prevBin = -1;
    int currBin;

    //go through each pixel in the tile
    for (int i = 0; i < TILEWIDTH; i++)
    {
        col = colStart + i;
        if (row < height && col < width)
        {
            //flatten the 2D indices
            int pIndx = row * pitch + col * CHANNELS;

            unsigned char redVal = Pin[pIndx];
            unsigned char greenVal = Pin[pIndx + 1];
            unsigned char blueVal = Pin[pIndx + 2];
            currBin = (redVal/TONESPB)*BINS*BINS + (blueVal/TONESPB)*BINS
                       + greenVal/TONESPB;
            if (currBin != prevBin)
            {
                if (accumulator > 0) 
                    atomicAdd(&(histo->histogram[prevBin]), accumulator); 
                prevBin = currBin;
                accumulator = 1;
            } else accumulator++;
        }
    }
    if (accumulator > 0)
    {
        atomicAdd(&(histo->histogram[prevBin]), accumulator); 
    }
}

/*
    classifyOnGPU
    Performs image classification on the GPU

    Outputs:
    comparisons - an array of size modelCt. comparisons[i] is set to the
                  result of comparing the input image to model i
                  The size of this array is modelCt.

    Inputs:
    histograms - an array of histograms. 
        The histogram for the input image is in:
        histograms[0] ... histogram[TOTALBINS - 1]
        The histogram for model 0 is in:
        histograms[TOTALBINS] ... histogram[2*TOTALBINS - 1]

        The histogram for the last model is in:
        histograms[modelCt*TOTALBINS] ... histogram[modelCt*TOTALBINS - 1]
        Thus, note that the array contains the input histogram and the
        model histograms and thus is of size (modelCt + 1) * TOTALBINS
   
    modelCt - count of the number of models used for the classification 
 
    Returns the amount of time it takes to classify the image
*/
float classifyOnGPU(float * comparisons, int * histograms, int modelCt)
{
    hipEvent_t start_gpu, stop_gpu;
    float gpuMsecTime = -1;

    //THIS FUNCTION IS NOT COMPLETE.  You need to:
    //1) allocate a float array on the GPU to hold the normalized histograms
    //It needs to be big enough to hold the histogram of the input image and
    //and the histograms of all of the models.
    float * normHistograms;

    //2) allocate an int array on the GPU to hold the original histograms
    //It needs to be big enough to hold the histogram of the input image and
    //and the histograms of all of the models.
    int * dhistograms;

    //3) copy input histograms into dhistograms
 
    //4) allocate a float array on the GPU to hold the comparisons
    //there needs to be one element per model
    float * dcomparisons;


    //THE REST OF THIS FUNCTION IS COMPLETE 
    //Use cuda functions to do the timing 
    //create event objects
    CHECK(hipEventCreate(&start_gpu));
    CHECK(hipEventCreate(&stop_gpu));

    //record the starting time
    CHECK(hipEventRecord(start_gpu));

    //each model is handled by a single block of threads
    //an extra block of threads is needed to normalize the input histogram
    dim3 grid(modelCt + 1, 1, 1);
    //don't make block any larger than the number of bins
    dim3 block(min(TOTALBINS, CLASSBLOCKDIM), 1);

    d_classifyKernel<<<grid, block>>>(dcomparisons, normHistograms, dhistograms);

    CHECK(hipEventRecord(stop_gpu));

    //copy the device comparison array into the host comparison array
    CHECK(hipMemcpy(comparisons, dcomparisons, sizeof(float) * modelCt,
          hipMemcpyDeviceToHost));

    //record the ending time and wait for event to complete
    CHECK(hipEventSynchronize(stop_gpu));
    //calculate the elapsed time between the two events 
    CHECK(hipEventElapsedTime(&gpuMsecTime, start_gpu, stop_gpu));

    return gpuMsecTime;
}

/*
    d_classifyKernel
    Kernel used to do the image classification on the GPU.  Each block of
    threads normalizes a single histogram. After that, every block except
    for block 0 will perform the intersection and store a
    result in the comparisons array.
    Thus, each block (except for 0) produces one result for the comparisons
    array.  Each thread in a block handles TOTALBINS/blockDim.x elements
 
    Inputs: 
    histograms - array of size gridDim.x * TOTALBINS. It contains
                 gridDim.x histograms each of size TOTALBINS.  The first one 
                 is the input histogram.
    Outputs:
    comparisons - comparison[i] is set to the value of the comparison of the
                  input histogram and the histogram of model i; for example,
                  comparison[0] is set to comparison of the input and model 0.
    normHistograms - array of size gridDim.x * TOTALBINS.  It contains
                     gridDim.x histograms that are equal to the normalization
                     of the input histograms.
*/

__device__ int blockSync = 0;   //need this to provide synchronization among blocks
__global__ void d_classifyKernel(float * comparisons, float * normHistograms, int * histograms) 
{
    __shared__ int histSz;
    __shared__ float intersect;

    //The device functions (that need to be written) are below this function

    //1) initialize histSz and intersect to 0

    //2) compute the size of the histogram handled by this block

    //3) normalize the histogram 

    //4) after block 0 has finished computing the normalized histogram,
    //one thread in its block should set blockSync to 1 so other blocks can
    //then proceed to compute the intersection

    //5) if not a block 0 thread, wait until blockSync is no longer 0 before 
    //continuing (page 193 has logic similar to what has to be done here)

    //6) compute the intersection

    //7) one thread in all blocks except 0 should store the fractional intersect
    //value in the comparisons array
}


/* 
    intersection
    Calculates the intersection of the input histogram and a model histogram
    after they have been normalized.
    The input histogram is in normHistograms[0] ... normHistograms[TOTALBINS - 1]
    The model histogram is in normHistograms[TOTALBINS * blockIdx.x] ...
    normHistograms[TOTALBINS * (blockIdx.x + 1) - 1]
   
    Inputs:
    normHistograms - array of TOTALBINS * gridDim.x bins (gridDim.x histograms)
    intersect - pointer to the shared intersect value

    Outputs:
    shared intersect variable is incremented by the intersection calculated by the
       thread running this code 
*/
__device__ void intersection(float * normHistograms, float * intersect)
{
    //compute intersection using cyclic partitioning

}

/*
    computeHistSz
    Calculates the size of a histogram by adding up all of the bin
    values. The histogram to be used for the calculation is in elements
    histograms[blockIdx.x * TOTALBINS] ... histograms[(blockIdx.x + 1) * TOTALBINS - 1]
  
    Inputs:
    histograms - array of TOTALBINS * gridDim.x bins (gridDim.x histograms)
    histSz - pointer to the shared histogram size variable

    Outputs:
    shared histogram size variable is incremented by the size calculated by
         the thread running this code
*/
__device__ void computeHistSz(int * histograms, int * histSz)
{
    //compute histogram size (sum of bins) using cyclic partitioning

}

/*
    normalizeHist
    Normalizes the histogram so that every bin value is between 0 and NORMMAX.
    The histogram to be normalized is in elements
    histograms[blockIdx.x * TOTALBINS] ... histograms[(blockIdx.x + 1) * TOTALBINS - 1]
    The result will be stored in normHistograms[blockIdx.x * TOTALBINS] ... 
    normHistograms[(blockIdx.x + 1) * TOTALBINS - 1]

    Inputs:
    histograms - array that holds the histogram to be normalized
    histSz - size of the input histogram (sum of its bins)

    Outputs:
    normHistograms - array to hold the normalized histogram
*/
__device__ void normalizeHist(float * normHistograms, int * histograms, int histSz)
{
    //compute the normalized histogram using cyclic partitioning

} 

//this can be used for debugging
__device__ void printFloatArray(float * array, int startIdx, int length)
{
    int i, j = 0;
    for (i = startIdx; i < startIdx + length; i++, j++)
    {
        if ((j % 16) == 0) printf("\n%3d: ", i);
        printf("%6.1f ", array[i]);
    } 
}        

//launched to get more accurate timing
__global__ void emptyKernel()
{
}
